#include "hip/hip_runtime.h"

#include <cogcuda/Sampler.cuh>

#include <hip/hip_runtime.h>
#include <>

namespace cognitics
{
    namespace cuda
    {
        bool Available()
        {
            int device_count;
            auto err = hipGetDeviceCount(&device_count);
            return (err == hipSuccess) && (device_count > 0);
        }

        namespace
        {
            __global__ void kernel_sample(SamplerRaster* input_array, size_t input_count, SamplerRaster* out)
            {
                int x = threadIdx.x + (blockIdx.x * blockDim.x);
                int y = threadIdx.y + (blockIdx.y * blockDim.y);
                double lat = out->South + (y * ((out->North - out->South) / out->Height));
                double lon = out->West + (x * ((out->East - out->West) / out->Width));

                // pick the first matching input raster
                // input_array should be sorted from highest resolution to lowest
                SamplerRaster* input = nullptr;
                for(size_t input_index = 0; input_index < input_count; ++input_index)
                {
                    if(lat <= input_array[input_index].South)
                        continue;
                    if(lat >= input_array[input_index].North)
                        continue;
                    if(lon <= input_array[input_index].West)
                        continue;
                    if(lon >= input_array[input_index].East)
                        continue;
                    input = &input_array[input_index];
                    break;
                }

                if(input == nullptr)
                    return;

                double spacing_x = (input->East - input->West) * input->Width;
                double spacing_y = (input->North - input->South) * input->Height;
                int input_x = std::floor((lon - input->West) / spacing_x);
                int input_y = std::floor((lat - input->South) / spacing_y);
                int sw_index = (y * input->Width) + x;
                double sw_lat = input->South + (input_y * spacing_y);
                double sw_lon = input->West + (input_x * spacing_x);

                float sw_value = input->Data[sw_index];
                float se_value = input->Data[sw_index + 1];
                float nw_value = input->Data[sw_index + input->Width];
                float ne_value = input->Data[sw_index + input->Width + 1];

                // bilinear interpolation
                float nx = (lon - sw_lon) / spacing_x;
                float ny = (lat - sw_lat) / spacing_y;
                float a00 = sw_value;
                float a10 = se_value - sw_value;
                float a01 = nw_value - sw_value;
                float a11 = sw_value - se_value - nw_value + ne_value;
                float value = a00 + (a10 * nx) + (a01 * ny) + (a11 * nx * ny);

                out->Data[(y * out->Width) + x] = value;
            }

        }

        Sampler::Sampler(const std::vector<SamplerRaster>& rasters)
        {
            // input must be sorted from highest resolution to lowest
            // all input rasters must contain valid width/height
            InputCount = rasters.size();
            hipMallocManaged(&InputArray, sizeof(SamplerRaster) * InputCount);
            for(size_t i = 0; i < InputCount; ++i)
            {
                auto& source = rasters[i];
                auto& input = InputArray[i];
                input = source;
                hipMallocManaged(&input.Data, input.Width * input.Height * sizeof(float));
                std::copy(source.Data, source.Data + (input.Width * input.Height), input.Data);
            }
        }

        void Sampler::Sample(SamplerRaster* output, float nodata)
        {
            // output must contain a valid width/height
            SamplerRaster* kernel_output;
            hipMallocManaged(&kernel_output, sizeof(SamplerRaster));
            *kernel_output = *output;
            hipMallocManaged(&kernel_output->Data, output->Width * output->Height * sizeof(float));
            for(size_t i = 0, c = output->Width * output->Height; i < c; ++i)
                kernel_output->Data[i] = nodata;
            dim3 block_size(16, 16); // block threads must be <= 1024 (32x32 max)
            dim3 block_count(kernel_output->Width / block_size.x, kernel_output->Height / block_size.y);
            kernel_sample<<<block_count, block_size>>>(InputArray, InputCount, kernel_output);
            hipDeviceSynchronize();
            for(size_t i = 0, c = output->Width * output->Height; i < c; ++i)
                output->Data[i] = kernel_output->Data[i];
            hipFree(kernel_output->Data);
            hipFree(kernel_output);
        }


    }
}
